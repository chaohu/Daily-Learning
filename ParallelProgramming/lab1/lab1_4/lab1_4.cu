#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

void addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main() {
	const int arraySize = 10;
	const int a[arraySize] = {0,1,2,3,4,5,6,7,8,9};
	const int b[arraySize] = {0,1,2,3,4,5,6,7,8,9};
	int c[arraySize] = {0,0,0,0,0,0,0,0,0,0};

	// Add vectors in parallel.
	addWithCuda(c, a, b, arraySize);
	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",c[0], c[1], c[2], c[3], c[4]);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, size_t size) {
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipSetDevice(0);
	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	printf("%d\n",cudaStatus);
	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipDeviceSynchronize();
	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}

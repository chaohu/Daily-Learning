#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2/opencv.hpp>
using namespace cv;

void conWithCuda(const Mat *img, Mat *result, size_t size);

__global__ void conKernel(const uchar *img, uchar *result, const int *_size, const int *_rows, const int *_cols) {
	int t = threadIdx.x;
	int i = 0,j = 0,k = 0,n = 3;
	int size = *_size;
	int rows = *_rows;
	int cols = *_cols;
	int start = 1 + t * (rows-2) / size;
	int end = (t+1 == size) ? rows-2 : (t+1)*(rows-2)/size;
	int temp = 0;

	for(i = start;i <= end;i++) {
		const uchar *previous = img+(i-1)*cols*3;
		const uchar *current = img+i*cols*3;
		const uchar *next = img+(i+1)*cols*3;
		uchar *output = result+i*cols*3;
		for(j = 1;j < cols-1;j++) {
			for(k = 0;k < n;k++) {
				//锐化操作
				//output[j*n+k] = saturate_cast<uchar>(9*current[j*n+k] - previous[(j-1)*n+k] - previous[j*n+k] - previous[(j+1)*n+k] - current[(j-1)*n+k] - current[(j+1)*n+k] - next[(j-1)*n+k] - next[j*n+k] - next[(j+1)*n+k]);
				//边缘操作
				temp = (-7)*current[j*n+k] + previous[(j-1)*n+k] + previous[j*n+k] + previous[(j+1)*n+k] + current[(j-1)*n+k] + current[(j+1)*n+k] + next[(j-1)*n+k] + next[j*n+k] + next[(j+1)*n+k];
				if(temp < 0) output[j*n+k] = 0;
				else if(temp > 255) output[j*n+k] = 255;
				else output[j*n+k] = (uchar)temp;
			}
		} 
	}
}

int main() {
	int size = 10;
	Mat img = imread("hehe.jpg");
	Mat result;

	if(img.empty()) {
		printf("open image failed\n");
		return -1;
	}
	result.create(img.size(),img.type());

	// converlution in parallel.
	conWithCuda(&img, &result, size);
	imwrite("pppp.jpg",result);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();
	return 0;
}

// Helper function for using CUDA to convolution in parallel.
void conWithCuda(const Mat *img, Mat *result, size_t size) {
	int *dev_size = 0;
	int *dev_rows = 0;
	int *dev_cols = 0;
	uchar *dev_img = 0;
	uchar *dev_result = 0;
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipSetDevice(0);
	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_size, sizeof(int));
	hipMalloc((void**)&dev_rows, sizeof(int));
	hipMalloc((void**)&dev_cols, sizeof(int));
	hipMalloc((void**)&dev_img, img->rows * img->cols * sizeof(uchar) * 3);
	hipMalloc((void**)&dev_result, img->rows * img->cols * sizeof(uchar) *3);
	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_size, &size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_rows, &(img->rows), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_cols, &(img->cols), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_img, img->ptr<uchar>(0), img->rows * img->cols * sizeof(uchar) * 3, hipMemcpyHostToDevice);
	// Launch a kernel on the GPU with one thread for each element.
	conKernel<<<1, size>>>(dev_img, dev_result, dev_size, dev_rows, dev_cols);
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipDeviceSynchronize();
	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(result->ptr<uchar>(0), dev_result, img->rows * img->cols * sizeof(uchar) * 3, hipMemcpyDeviceToHost);
	hipFree(dev_size);
	hipFree(dev_rows);
	hipFree(dev_cols);
	hipFree(dev_img);
	hipFree(dev_result);
}
